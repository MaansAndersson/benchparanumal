#include "hip/hip_runtime.h"

#define p_G00ID 0
#define p_G01ID 1
#define p_G02ID 2
#define p_G11ID 3
#define p_G12ID 4
#define p_G22ID 5


#define p_Nq (p_N+1)
#define p_cubNq (p_N+2)
#define p_Np ( p_Nq*p_Nq*p_Nq )
#define p_cubNp ( p_cubNq*p_cubNq*p_cubNq )

__global__ void benchpBP3Global_v0(const int Nelements,
				   void *context,
				   CudaFieldsInt * __restrict__ localizedIds,
				   const double * __restrict__ q,
				   const double * __restrict__ ggeo,
				   const double * __restrict__ I,
				   const double * __restrict__ D,
				   double * __restrict__ Aq){
  
  int e = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  __shared__ double s_Iq[p_cubNq][p_cubNq][p_cubNq];	
  
  __shared__ double s_D[p_cubNq][p_cubNq];
  __shared__ double s_I[p_cubNq][p_Nq];
  
  __shared__ double s_Gqr[p_cubNq][p_cubNq];
  __shared__ double s_Gqs[p_cubNq][p_cubNq];
  
  double r_qt, r_q[p_cubNq], r_Aq[p_cubNq];
  
  // array of threads
  s_D[ty][tx] = D[p_cubNq*ty+tx]; 
  
  if(tx<p_Nq){
    s_I[ty][tx] = I[p_Nq*ty+tx];
  }
  
  // load pencil of u into register
  if(tx<p_Nq && ty<p_Nq){
    for(int k = 0; k < p_Nq; k++) {
      const int id = e*p_Np + k*p_Nq*p_Nq+ ty*p_Nq + tx;
      int localId = localizedIds[id]-1;
      r_q[k] = q[localId];
    }
  }

  __syncthreads();
  
  {
    int b = ty, a = tx;
    if(a<p_Nq && b<p_Nq){				
      for(int k=0;k<p_cubNq;++k){			
	double res = 0;				
	for(int c=0;c<p_Nq;++c){			
	  res += s_I[k][c]*r_q[c];			
	}						
	s_Iq[k][b][a] = res;			
      }						
    }
  }

  __syncthreads();

  {
    int k = ty, a = tx;
    if(a<p_Nq){					
      for(int b=0;b<p_Nq;++b){			
	r_Aq[b] = s_Iq[k][b][a];			
      }						
      
      for(int j=0;j<p_cubNq;++j){			
	double res = 0;				
	for(int b=0;b<p_Nq;++b){			
	  res += s_I[j][b]*r_Aq[b];			
	}						
	s_Iq[k][j][a] = res;			
      }						
    }						
  }

  __syncthreads();
  
  {
    int k = ty, j = tx;
    for(int a=0;a<p_Nq;++a){			
      r_Aq[a] = s_Iq[k][j][a];			
    }						
    
    for(int i=0;i<p_cubNq;++i){			
      double res = 0;				
      for(int a=0;a<p_Nq;++a){			
	res += s_I[i][a]*r_Aq[a];			
      }						
      s_Iq[k][j][i] = res;				
    }						
  }							
  
  {
    for(int k = 0; k < p_cubNq; k++) {
      r_Aq[k] = 0.f; // zero the accumulator
    }
  }
  
  // Layer by layer
#pragma unroll p_cubNq
  for(int k = 0;k < p_cubNq; k++){
    
    __syncthreads();
    
    int j = ty, i = tx;
      
    // share u(:,:,k)
    double qr = 0, qs = 0;
      
    r_qt = 0;
      
#pragma unroll p_cubNq
    for(int m = 0; m < p_cubNq; m++) {
      double Dim = s_D[i][m];
      double Djm = s_D[j][m];
      double Dkm = s_D[k][m];
	
      qr += Dim*s_Iq[k][j][m];
      qs += Djm*s_Iq[k][m][i];
      r_qt += Dkm*s_Iq[m][j][i];	    
    }
      
    // prefetch geometric factors
    //    const int gbase = e*p_Nggeo*p_cubNp + k*p_cubNq*p_cubNq + j*p_cubNq + i;
    const int gbase = e*p_cubNp + k*p_cubNq*p_cubNq + j*p_cubNq + i;
    const int stride = p_cubNp*Nelements;

    const double G00 = ggeo[gbase+p_G00ID*stride];
    const double G01 = ggeo[gbase+p_G01ID*stride];
    const double G02 = ggeo[gbase+p_G02ID*stride];
    const double G11 = ggeo[gbase+p_G11ID*stride];
    const double G12 = ggeo[gbase+p_G12ID*stride];
    const double G22 = ggeo[gbase+p_G22ID*stride];
      
    s_Gqr[j][i] = (G00*qr + G01*qs + G02*r_qt);
    s_Gqs[j][i] = (G01*qr + G11*qs + G12*r_qt);
      
    r_qt = G02*qr + G12*qs + G22*r_qt;
      
    __syncthreads();
	  
    double Aqtmp = 0;
      
#pragma unroll p_cubNq
    for(int m = 0; m < p_cubNq; m++){
      double Dmi = s_D[m][i];
      double Dmj = s_D[m][j];
      double Dkm = s_D[k][m];
	
      Aqtmp += Dmi*s_Gqr[j][m];
      Aqtmp += Dmj*s_Gqs[m][i];
      r_Aq[m] += Dkm*r_qt;
    }
      
    r_Aq[k] += Aqtmp;
  }

  __syncthreads();

  {							
    /* lower 'k' */
    {
      int j = ty, i = tx;
      							
      for(int c=0;c<p_Nq;++c){			
	double res = 0;				
	for(int k=0;k<p_cubNq;++k){			
	  res += s_I[k][c]*r_q[k];			
	}						
	s_Iq[c][j][i] = res;				
      }						
    }

    __syncthreads();
      
    {
      int c = ty, i = tx;
							
      if(c<p_Nq){					
	for(int j=0;j<p_cubNq;++j){			
	  r_q[j] = s_Iq[c][j][i];			
	}						
	  
	for(int b=0;b<p_Nq;++b){			
	  double res = 0;				
	  for(int j=0;j<p_cubNq;++j){			
	    res += s_I[j][b]*r_q[j];			
	  }						
	    
	  s_Iq[c][b][i] = res;			
	}						
      }						
    }

    __syncthreads();

    {
      int c = ty, b = tx;
							
      if(b<p_Nq && c<p_Nq){				
	for(int i=0;i<p_cubNq;++i){			
	  r_q[i] = s_Iq[c][b][i];			
	}						
	  
	for(int a=0;a<p_Nq;++a){			
	  double res = 0;				
	  for(int i=0;i<p_cubNq;++i){			
	    res += s_I[i][a]*r_q[i];			
	  }						
	    
	  s_Iq[c][b][a] = res;			
	}						
      }						
    }							
  }
    
  // write out

  {
    int j = ty, i = tx;
    if(i<p_Nq && j<p_Nq){
#pragma unroll p_Nq
      for(int k = 0; k < p_Nq; k++){
	const int id = e*p_Np +k*p_Nq*p_Nq+ j*p_Nq + i;
	int localId = localizedIds[id]-1;
	double res = s_Iq[k][j][i];
	atomicAdd(Aq+localId, res); // atomic assumes Aq zerod
      }
    }
  }
}
}

#if p_cubNq==3
#define p_Nblk 3
#elif p_cubNq==4
#define p_Nblk 2
#else
#define p_Nblk 1
#endif

__global__ void BP3Global_v1(const int Nelements,
			     void *context,
			     CudaFieldsInt * __restrict__ localizedIds,
			     const double * __restrict__ q,
			     const double * __restrict__ ggeo,
			     const double * __restrict__ I,
			     const double * __restrict__ D,
			     double * __restrict__ Aq){
  
  int eo = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int es = threadIdx.z;
    
  __shared__ double s_Iq[p_Nblk][p_cubNq][p_cubNq][p_cubNq];	
    
  __shared__ double s_D[p_cubNq][p_cubNq];
  __shared__ double s_I[p_cubNq][p_Nq];
    
  __shared__ double s_Gqr[p_Nblk][p_cubNq][p_cubNq];
  __shared__ double s_Gqs[p_Nblk][p_cubNq][p_cubNq];

  double r_qt;
    
  // heavy on registers (FP64, 2*3*8 for N=7)
  double r_q[p_cubNq], r_Aq[p_cubNq];

  int r_e = eo + es;

  {
    int j = ty, i = tx;
      
    if(es==0){
      s_D[j][i] = D[p_cubNq*j+i]; 
	
      if(i<p_Nq){
	s_I[j][i] = I[p_Nq*j+i];
      }
    }
      
    if(r_e<Nelements){
      // load pencil of u into register
	
      if(i<p_Nq && j<p_Nq){
	for(int k = 0; k < p_Nq; k++) {
	  const int id = r_e*p_Np +k*p_Nq*p_Nq+ j*p_Nq + i;
	  int localId = localizedIds[id]-1;
	  r_q[k] = q[localId];
	}
      }
    }
  }

  __syncthreads();
    
  {
    int b = ty, a = tx;
    if(a<p_Nq && b<p_Nq){
#pragma unroll p_cubNq
      for(int k=0;k<p_cubNq;++k){			
	double res = 0;
#pragma unroll p_Nq
	for(int c=0;c<p_Nq;++c){			
	  res += s_I[k][c]*r_q[c];			
	}						
	s_Iq[es][k][b][a] = res;			
      }							
    }							
  }

  __syncthreads();

  {
    int k = ty, a = tx;
							
    if(a<p_Nq){					
      for(int b=0;b<p_Nq;++b){			
	r_Aq[b] = s_Iq[es][k][b][a];		
      }						
	
#pragma unroll p_cubNq
      for(int j=0;j<p_cubNq;++j){			
	double res = 0;
#pragma unroll p_Nq
	for(int b=0;b<p_Nq;++b){			
	  res += s_I[j][b]*r_Aq[b];			
	}						
	s_Iq[es][k][j][a] = res;			
      }						
    }						
  }

  __syncthreads();
    
  {
    int k = ty, j = tx;
    for(int a=0;a<p_Nq;++a){			
      r_Aq[a] = s_Iq[es][k][j][a];			
    }						
      
#pragma unroll p_cubNq
    for(int i=0;i<p_cubNq;++i){			
      double res = 0;
#pragma unroll p_Nq
      for(int a=0;a<p_Nq;++a){			
	res += s_I[i][a]*r_Aq[a];			
      }						
      s_Iq[es][k][j][i] = res;			
    }						
      
    for(int a = 0; a < p_cubNq; a++) {
      r_Aq[a] = 0.f; // zero the accumulator
    }
  }

  // Layer by layer
#pragma unroll p_cubNq
  for(int k = 0;k < p_cubNq; k++){
      
    __syncthreads();

    {
      int j = ty, i = tx;

      if(r_e<Nelements){
	double qr = 0, qs = 0;
	  
	r_qt = 0;
	  
#pragma unroll p_cubNq
	for(int m = 0; m < p_cubNq; m++) {
	  double Dim = s_D[i][m];
	  double Djm = s_D[j][m];
	  double Dkm = s_D[k][m];
	    
	  qr += Dim*s_Iq[es][k][j][m];
	  qs += Djm*s_Iq[es][k][m][i];
	  r_qt += Dkm*s_Iq[es][m][j][i];	    
	}
	  
	// prefetch geometric factors
	const int gbase = r_e*p_cubNp + k*p_cubNq*p_cubNq + j*p_cubNq + i;
	const int stride = p_cubNp*Nelements;
	
	const double G00 = ggeo[gbase+p_G00ID*stride];
	const double G01 = ggeo[gbase+p_G01ID*stride];
	const double G02 = ggeo[gbase+p_G02ID*stride];
	const double G11 = ggeo[gbase+p_G11ID*stride];
	const double G12 = ggeo[gbase+p_G12ID*stride];
	const double G22 = ggeo[gbase+p_G22ID*stride];
	  
	s_Gqr[es][j][i] = (G00*qr + G01*qs + G02*r_qt);
	s_Gqs[es][j][i] = (G01*qr + G11*qs + G12*r_qt);
	  
	r_qt = G02*qr + G12*qs + G22*r_qt;
	  
      }
    }

    __syncthreads();
      
    {
      int j = ty, i = tx;
	    
      double Aqtmp = 0;
	
#pragma unroll p_cubNq
      for(int m = 0; m < p_cubNq; m++){
	double Dmi = s_D[m][i];
	double Dmj = s_D[m][j];
	double Dkm = s_D[k][m];
	  
	Aqtmp += Dmi*s_Gqr[es][j][m];
	Aqtmp += Dmj*s_Gqs[es][m][i];
	r_Aq[m] += Dkm*r_qt;
      }
	
      r_Aq[k] += Aqtmp;
    }
      
    __syncthreads();
      
    {
      int j = ty, i = tx;

#pragma unroll p_Nq	    
      for(int c=0;c<p_Nq;++c){		
	double res = 0;
#pragma unroll p_cubNq
	for(int k=0;k<p_cubNq;++k){		
	  res += s_I[k][c]*r_Aq[k];		
	}					
	s_Iq[es][c][j][i] = res;		
      }						
    }

    __syncthreads();

    {
      int c = ty, i = tx;
						
      if(c<p_Nq){				
	for(int j=0;j<p_cubNq;++j){		
	  r_q[j] = s_Iq[es][c][j][i];	
	}					
	  
#pragma unroll p_Nq	    
	for(int b=0;b<p_Nq;++b){		
	  double res = 0;
#pragma unroll p_cubNq	      
	  for(int j=0;j<p_cubNq;++j){	
	    res += s_I[j][b]*r_q[j];	
	  }					
	    
	  s_Iq[es][c][b][i] = res;		
	}					
      }					
    }

    __syncthreads();

    {
      int c = ty, b = tx;
						
      if(b<p_Nq && c<p_Nq){			
	for(int i=0;i<p_cubNq;++i){		
	  r_q[i] = s_Iq[es][c][b][i];	
	}					
	  
#pragma unroll p_Nq	    
	for(int a=0;a<p_Nq;++a){		
	  double res = 0;
#pragma unroll p_cubNq
	  for(int i=0;i<p_cubNq;++i){	
	    res += s_I[i][a]*r_q[i];	
	  }					
	    
	  s_Iq[es][c][b][a] = res;		
	}					
      }
    }
  }						
    
  __syncthreads();
    
  {
    int j = ty, i = tx;
    if(r_e<Nelements){
      if(i<p_Nq && j<p_Nq){
#pragma unroll p_Nq
	for(int k = 0; k < p_Nq; k++){
	  const int id = r_e*p_Np +k*p_Nq*p_Nq+ j*p_Nq + i;
	  int localId = localizedIds[id]-1;
	  double res  = s_Iq[es][k][j][i];
	  atomicAdd(Aq+localId, res);
	}
      }
    }
  }
}

#if 0
__global__ void BP3Global_v2(const int Nelements,
			     @restrict const int  *elementList,
			     @restrict const int *localizedIds,
			     @restrict const double *ggeo,
			     @restrict const double *D,
			     @restrict const double *I,
			     const double lambda,
			     @restrict const double *q,
			     @restrict double *Aq){
  
  for(int e=0; e<Nelements; ++e; @outer(0)){
    
    __shared__ double s_Iq[p_cubNq][p_cubNq][p_cubNq];	
    
    __shared__ double s_D[p_cubNq][p_cubNq];
    __shared__ double s_I[p_cubNq][p_Nq];
    
    __shared__ double s_Gqr[p_cubNq][p_cubNq];
    __shared__ double s_Gqs[p_cubNq][p_cubNq];

    @exclusive double r_qt;
    
    // heavy on registers (FP64, 2*3*8 for N=7)
    @exclusive double r_q[p_cubNq];

    @exclusive int element;
    
    // array of threads
    for(int j=0;j<p_cubNq;++j;@inner(1)){
      for(int i=0;i<p_cubNq;++i;@inner(0)){
	
        s_D[j][i] = D[p_cubNq*j+i]; 
	
	if(i<p_Nq){
	  s_I[j][i] = I[p_Nq*j+i];
	}

	element = elementList[e];
	
        // load pencil of u into register
	if(i<p_Nq && j<p_Nq){
	  for(int k = 0; k < p_Nq; k++) {
	    const int id = element*p_Np +k*p_Nq*p_Nq+ j*p_Nq + i;
	    int localId = localizedIds[id]-1;
	    r_q[k] = q[localId];
	  }
	}
      }
    }
    
    // raise pressure degree
    //    interpolateHex3D(s_I, r_q, s_Iq);
    for(int b=0;b<p_cubNq;++b;@inner(1)){		
      for(int a=0;a<p_cubNq;++a;@inner(0)){		
	if(a<p_Nq && b<p_Nq){				
	  for(int k=0;k<p_cubNq;++k){			
	    double res = 0;				
	    for(int c=0;c<p_Nq;++c){			
	      res += s_I[k][c]*r_q[c];			
	    }						
	    s_Iq[k][b][a] = res;			
	  }						
	}						
      }							
    }							
    for(int k=0;k<p_cubNq;++k;@inner(1)){		
      for(int a=0;a<p_cubNq;++a;@inner(0)){		
							
	if(a<p_Nq){					
	  for(int b=0;b<p_Nq;++b){			
	    r_q[b] = s_Iq[k][b][a];			
	  }						
	  						
	  for(int j=0;j<p_cubNq;++j){			
	    double res = 0;				
	    for(int b=0;b<p_Nq;++b){			
	      res += s_I[j][b]*r_q[b];			
	    }						
	    s_Iq[k][j][a] = res;			
	  }						
	}						
      }							
    }							
    for(int k=0;k<p_cubNq;++k;@inner(1)){		
      for(int j=0;j<p_cubNq;++j;@inner(0)){		
	for(int a=0;a<p_Nq;++a){			
	  r_q[a] = s_Iq[k][j][a];			
	}						
							
	for(int i=0;i<p_cubNq;++i){			
	  double res = 0;				
	  for(int a=0;a<p_Nq;++a){			
	    res += s_I[i][a]*r_q[a];			
	  }						
	  s_Iq[k][j][i] = res;				
	}						

	for(int a = 0; a < p_cubNq; a++) {
	  r_q[a] = 0.f; // zero the accumulator
	}
      }
    }
    
    // Layer by layer
#pragma unroll p_cubNq
    for(int k = 0;k < p_cubNq; k++){
      
      @barrier("local");
      
      for(int j=0;j<p_cubNq;++j;@inner(1)){
        for(int i=0;i<p_cubNq;++i;@inner(0)){
	  
          // share u(:,:,k)
          double qr = 0, qs = 0;

	  r_qt = 0;
	  
#pragma unroll p_cubNq
          for(int m = 0; m < p_cubNq; m++) {
            double Dim = s_D[i][m];
            double Djm = s_D[j][m];
            double Dkm = s_D[k][m];

            qr += Dim*s_Iq[k][j][m];
            qs += Djm*s_Iq[k][m][i];
	    r_qt += Dkm*s_Iq[m][j][i];	    
          }
	  
          // prefetch geometric factors
          const int gbase = element*p_cubNp + k*p_cubNq*p_cubNq + j*p_cubNq + i;
	  const int stride = p_cubNp*Nelements;
	  
          const double G00 = ggeo[gbase+p_G00ID*stride];
          const double G01 = ggeo[gbase+p_G01ID*stride];
          const double G02 = ggeo[gbase+p_G02ID*stride];
          const double G11 = ggeo[gbase+p_G11ID*stride];
          const double G12 = ggeo[gbase+p_G12ID*stride];
          const double G22 = ggeo[gbase+p_G22ID*stride];
	  
          s_Gqr[j][i] = (G00*qr + G01*qs + G02*r_qt);
          s_Gqs[j][i] = (G01*qr + G11*qs + G12*r_qt);

          r_qt = G02*qr + G12*qs + G22*r_qt;

        }
      }
      
      for(int j=0;j<p_cubNq;++j;@inner(1)){
        for(int i=0;i<p_cubNq;++i;@inner(0)){
	  
	  double Aqtmp = 0;
	  
#pragma unroll p_cubNq
          for(int m = 0; m < p_cubNq; m++){
            double Dmi = s_D[m][i];
            double Dmj = s_D[m][j];
            double Dkm = s_D[k][m];
	    
            Aqtmp += Dmi*s_Gqr[j][m];
            Aqtmp += Dmj*s_Gqs[m][i];
            r_q[m] += Dkm*r_qt;
          }

          r_q[k] += Aqtmp;
        }
      }
    }
    
    // lower pressure degree
    testHex3D(s_I, r_q, s_Iq);

    // write out
    
    for(int j=0;j<p_cubNq;++j;@inner(1)){
      for(int i=0;i<p_cubNq;++i;@inner(0)){

	if(i<p_Nq && j<p_Nq){
#pragma unroll p_Nq
	  for(int k = 0; k < p_Nq; k++){
	    const int id = element*p_Np +k*p_Nq*p_Nq+ j*p_Nq + i;
	    int localId = localizedIds[id]-1;
	    double res = s_Iq[k][j][i];
	    atomicAdd(Aq+localId, res); // atomic assumes Aq zerod
	  }
	}
      }
    }
  }
}







#if 0
// assume ggeo encodes built blocks
__global__ void BP3Global_v2(const int Nelements,
			     @restrict const int  *elementList,
			     @restrict const int *localizedIds,
			     @restrict const double *ggeo,
			     @restrict const double *D,
			     @restrict const double *I,
			     const double lambda,
			     @restrict const double *q,
			     @restrict double *Aq){
  
  for(int e=0; e<Nelements; ++e; @outer(0)){

    __shared__ double s_q[p_Np];
    
    for(int n=0;n<p_Np;++n;@inner(0)){
      s_q[n] = q[n + e*p_Np];
    }

    
    for(int n=0;n<p_Np;++n;@inner(0)){
      double res = 0;
      const double *base = ggeo + e*p_Np*p_Np; // only works for N=1
      for(int m=0;m<p_Np;++m){
	res += base[m*p_Np+n]*s_q[m];
      }
      
      Aq[e*p_Np+n] = res;
    }
  }
}
    

// assume ggeo encodes built blocks
__global__ void BP3Dot_v2(const int Nelements,
			  @restrict const int  *elementList,
			  @restrict const int *localizedIds,
			  @restrict const double *ggeo,
			  @restrict const double *D,
			  @restrict const double *I,
			  const double lambda,
			  @restrict const double *q,
			  @restrict double *Aq,
			  @restrict double *qAq){
  
  for(int e=0; e<Nelements; ++e; @outer(0)){

    __shared__ double s_q[p_Np];
    __shared__ volatile double s_qAq[p_Np];
    __shared__ volatile double s_warp[32];
    
    for(int n=0;n<p_Np;++n;@inner(0)){
      s_q[n] = q[n + e*p_Np];
    }

    
    for(int n=0;n<p_Np;++n;@inner(0)){
      double res = 0;
      const double *base = ggeo + e*p_Np*p_Np; // only works for N=1
      for(int m=0;m<p_Np;++m){
	res += base[m*p_Np+n]*s_q[m];
      }
      
      Aq[e*p_Np+n] = res;

      s_qAq[n] = s_q[n]*res;
    }
    
    // do partial reduction on p.Ap [ two phase vSIMD32 sync ]
    for(int t=0;t<p_Np;++t;@inner(0)){
      int n = t%32;
      int w = t/32;
      
      // totally hard wired for SIMD32
      if(n<16 && t+16<p_Np) s_qAq[t] += s_qAq[t+16];
      if(n< 8 && t+8<p_Np)  s_qAq[t] += s_qAq[t+8];
      if(n< 4 && t+4<p_Np)  s_qAq[t] += s_qAq[t+4];
      if(n< 2 && t+2<p_Np)  s_qAq[t] += s_qAq[t+2];
      if(n< 1 && t+1<p_Np)  s_qAq[t] += s_qAq[t+1];
      if(n==0) s_warp[w] = s_qAq[t];
    }
    
    for(int t=0;t<p_Np;++t;@inner(0)){
      int n = t%32;
      int w = t/32;
      
      if(w==0 && n*32<p_Np){ // is this the base warp, and was there an entry from above
	if( n<16 && ((n+16)*32)<p_Np) s_warp[n] += s_warp[n+16];
	if( n< 8 && ((n+ 8)*32)<p_Np) s_warp[n] += s_warp[n+ 8];
	if( n< 4 && ((n+ 4)*32)<p_Np) s_warp[n] += s_warp[n+ 4];
	if( n< 2 && ((n+ 2)*32)<p_Np) s_warp[n] += s_warp[n+ 2];
	if( n< 1 && ((n+ 1)*32)<p_Np) s_warp[n] += s_warp[n+ 1];
	
	if(n==0){
	  double res = s_warp[0];
	  atomicAdd(qAq, res);
	}
      }
    }       
  }
}

__global__ void BP3Global_v2(const int Nelements,
			     @restrict const int  *elementList,
			     @restrict const int *localizedIds,
			     @restrict const double *ggeo,
			     @restrict const double *D,
			     @restrict const double *I,
			     const double lambda,
			     @restrict const double *q,
			     @restrict double *Aq){
  
  for(int eo=0; eo<Nelements; eo+=p_Nblk; @outer(0)){

    __shared__ double s_q[p_Nblk][p_Np];
    @exclusive int r_e, element;
    
    for(int es=0;es<p_Nblk;++es;@inner(1)){
      for(int n=0;n<p_Np;++n;@inner(0)){
	r_e = es + eo;
	if(r_e<Nelements){
	  element = elementList[r_e];
	  s_q[es][n] = q[n + element*p_Np];
	}
      }
    }
    
    for(int es=0;es<p_Nblk;++es;@inner(1)){    
      for(int n=0;n<p_Np;++n;@inner(0)){
	if(r_e<Nelements){
	  double res = 0;
	  const double *base = ggeo + element*p_Np*p_Np; // only works for N=1 (ggeo needs to be hijacked to make this work)
	  for(int m=0;m<p_Np;++m){
	    res += base[m*p_Np+n]*s_q[es][m];
	  }
	  
	  Aq[element*p_Np+n] = res;
	}
      }
    }
  }
}

#endif
#endif
